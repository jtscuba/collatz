/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

__global__ void
verifyCollatz(int64_t maxNumber)
{
	int timesToRunGrid = maxNumber / (blockDim.x * gridDim.x) + 1;

	int64_t number = 0;
	int64_t i = 0;

	for (int64_t gridRunNumber = 0; gridRunNumber < timesToRunGrid; ++gridRunNumber) {
		// odd numbers only
		number = 2 * (blockDim.x * gridDim.x * gridRunNumber + blockDim.x * blockIdx.x + threadIdx.x) + 1;
		i = number;
		if (number > 2 && number < maxNumber) {
			while (i >= number) {
				if (i & 0x1) {
					/* odd case */
					i = i * 3 + 1;
				} else {
					/* even case */
					i = i >> 1;
				}
			}
		}
	}
}

/**
 * Host main routine
 */
int
main()
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    int64_t maxNumber = 256ll * 256ll * 256ll * 256ll;


    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = 256;

	// use CUDA builtin heruistics to get max performance
	hipOccupancyMaxPotentialBlockSize(
        &blocksPerGrid,
        &threadsPerBlock,
        (void*) verifyCollatz,
        0, 0);
	
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	verifyCollatz<<<blocksPerGrid, threadsPerBlock>>>(maxNumber);
	err = hipGetLastError();

	hipDeviceSynchronize();
	
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch collatz  kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    printf("Done\n");
    return 0;
}

